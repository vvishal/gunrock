#include "hip/hip_runtime.h"
// ----------------------------------------------------------------
// Gunrock -- Fast and Efficient GPU Graph Library
// ----------------------------------------------------------------
// This source code is distributed under the terms of LICENSE.TXT
// in the root directory of this source distribution.
// ----------------------------------------------------------------

/**
 * @file
 * test_bfs.cu
 *
 * @brief Simple test driver program for breadth-first search.
 */

#include <stdio.h>
#include <string>
#include <deque>
#include <vector>
#include <iostream>

// Utilities and correctness-checking
#include <gunrock/util/test_utils.cuh>

// Graph construction utils
#include <gunrock/graphio/market.cuh>
#include <gunrock/graphio/rmat.cuh>
#include <gunrock/graphio/rgg.cuh>

// BFS includes
#include <gunrock/app/bfs/bfs_enactor.cuh>
#include <gunrock/app/bfs/bfs_problem.cuh>
#include <gunrock/app/bfs/bfs_functor.cuh>

// Operator includes
#include <gunrock/oprtr/advance/kernel.cuh>
#include <gunrock/oprtr/filter/kernel.cuh>

#include <moderngpu.cuh>

using namespace gunrock;
using namespace gunrock::util;
using namespace gunrock::oprtr;
using namespace gunrock::app::bfs;

/******************************************************************************
 * Switch 32/64-bit variables
 ******************************************************************************/
#define USE64 1

/******************************************************************************
 * Defines, constants, globals
 ******************************************************************************/

//bool g_verbose;
//bool g_undirected;
//bool g_quick;
//bool g_stream_from_host;

/******************************************************************************
 * Housekeeping Routines
 ******************************************************************************/
 void Usage()
 {
    printf(
        " test_bfs <graph type> <graph type args> [--device=<device_index>]\n"
        " [--undirected] [--instrumented] [--src=<source index>] [--quick=<0|1>]\n"
        " [--mark-pred] [--queue-sizing=<scale factor>] [--iteration-num=<num>]\n"
        " [--in-sizing=<in/out queue scale factor>] [--disable-size-check]\n "
        " [--grid-size=<grid size>] [partition_method=<random|biasrandom|clustered|metis]\n"
        " [--v] [--idempotence=<0|1>]\n"
        "\n"
        "Graph types and args:\n"
        "  market <file>\n"
        "    Reads a Matrix-Market coordinate-formatted graph of directed / undirected\n"
        "    edges from stdin (or from the optionally-specified file).\n"
        "  --device=<device_index>   Set GPU device for running the test. [Default: 0].\n"
        "  --undirected              Treat the graph as undirected (symmetric).\n"
        "  --idempotence=<0 or 1>    Enable: 1, Disable: 0 [Default: Enable].\n"
        "  --instrumented            Keep kernels statics [Default: Disable].\n"
        "                            total_queued, search_depth and barrier duty\n"
        "                            (a relative indicator of load imbalance.)\n"
        "  --src=<source vertex id>  Begins BFS from the source [Default: 0].\n"
        "                            If randomize: from a random source vertex.\n"
        "                            If largestdegree: from largest degree vertex.\n"
        "  --quick=<0 or 1>          Skip the CPU validation: 1, or not: 0 [Default: 1].\n"
        "  --mark-pred               Keep both label info and predecessor info.\n"
        "  --queue-sizing=<factor>   Allocates a frontier queue sized at: \n"
        "                            (graph-edges * <scale factor>). [Default: 1.0]\n"
        "  --v                       Print verbose per iteration debug info.\n"
        "  --iteration-num=<number>  Number of runs to perform the test [Default: 1].\n"
        "  --traversal-mode=<0 or 1> Set traversal strategy, 0 for Load-Balanced, \n"
        "                            1 for Dynamic-Cooperative [Default: dynamic\n"
        "                            determine based on average degree].\n"
        );
}

/**
 * @brief Displays the BFS result (i.e., distance from source)
 *
 * @param[in] source_path Search depth from the source for each node.
 * @param[in] preds Predecessor node id for each node.
 * @param[in] nodes Number of nodes in the graph.
 * @param[in] MARK_PREDECESSORS Whether to show predecessor of each node.
 * @param[in] ENABLE_IDEMPOTENCE Whether to enable idempotence mode.
 */
template<typename VertexId, typename SizeT, bool MARK_PREDECESSORS, bool ENABLE_IDEMPOTENCE>
void DisplaySolution(
    VertexId *labels,
    VertexId *preds,
    SizeT     num_nodes)
{
    if (num_nodes > 40) num_nodes = 40;

    printf("\nFirst %d labels of the GPU result:\n", num_nodes);

    printf("[");
    for (VertexId i = 0; i < num_nodes; ++i)
    {
        PrintValue(i);
        printf(":");
        PrintValue(labels[i]);
        if (MARK_PREDECESSORS && !ENABLE_IDEMPOTENCE)
        {
            printf(",");
            PrintValue(preds[i]);
        }
        printf(" ");
    }
    printf("]\n");
}

/**
 * Performance/Evaluation statistics
 */
struct Stats
{
    const char *name;
    Statistic rate;
    Statistic search_depth;
    Statistic redundant_work;
    Statistic duty;

    Stats() : name(NULL), rate(), search_depth(), redundant_work(), duty() {}
    Stats(const char *name) : name(name), rate(), search_depth(), redundant_work(), duty() {}
};

struct Test_Parameter : gunrock::app::TestParameter_Base {
public:
    bool          mark_predecessors ;// Whether or not to mark src-distance vs. parent vertices
    bool          enable_idempotence;// Whether or not to enable idempotence operation
    double        max_queue_sizing1 ;

    Test_Parameter()
    {
        mark_predecessors  = false;
        enable_idempotence = false;
        max_queue_sizing1  = -1.0;
    }

    ~Test_Parameter()
    {
    }

    void Init(CommandLineArgs &args)
    {
        TestParameter_Base::Init(args);
        mark_predecessors  = args.CheckCmdLineFlag("mark-pred");
        enable_idempotence = args.CheckCmdLineFlag("idempotence");
        args.GetCmdLineArgument("queue-sizing1", max_queue_sizing1);
    }
};

/**
 * @brief Displays timing and correctness statistics
 *
 * @tparam MARK_PREDECESSORS
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 *
 * @param[in] stats Reference to the Stats object defined in RunTests
 * @param[in] src Source node where BFS starts
 * @param[in] h_labels Host-side vector stores computed labels for validation
 * @param[in] graph Reference to the CSR graph we process on
 * @param[in] elapsed Total elapsed kernel running time
 * @param[in] search_depth Maximum search depth of the BFS algorithm
 * @param[in] total_queued Total element queued in BFS kernel running process
 * @param[in] avg_duty Average duty of the BFS kernels
 */
template<
    bool MARK_PREDECESSORS,
    typename VertexId,
    typename Value,
    typename SizeT>
void DisplayStats(
    Stats               &stats,
    VertexId            src,
    VertexId            *h_labels,
    const Csr<VertexId, Value, SizeT> *graph,
    double              elapsed,
    VertexId            search_depth,
    long long           total_queued,
    double              avg_duty)
{
    // Compute nodes and edges visited
    SizeT edges_visited = 0;
    SizeT nodes_visited = 0;
    for (VertexId i = 0; i < graph->nodes; ++i) {
        if (h_labels[i] < util::MaxValue<VertexId>() && h_labels[i]!=-1) {
            ++nodes_visited;
            edges_visited += graph->row_offsets[i+1] - graph->row_offsets[i];
        }
    }

    double redundant_work = 0.0;
    if (total_queued > 0)
    {
        // measure duplicate edges put through queue
        redundant_work = ((double)total_queued - edges_visited) / edges_visited;
    }
    redundant_work *= 100;

    // Display test name
    printf("[%s] finished. ", stats.name);

    // Display statistics
    if (nodes_visited < 5)
    {
        printf("Fewer than 5 vertices visited.\n");
    }
    else
    {
        // Display the specific sample statistics
        double m_teps = (double) edges_visited / (elapsed * 1000.0);
        printf("\n elapsed: %.4f ms, rate: %.4f MiEdges/s", elapsed, m_teps);
        if (search_depth != 0)
            printf(", search_depth: %lld", (long long) search_depth);
        if (avg_duty != 0)
        {
            printf("\n avg CTA duty: %.2f%%", avg_duty * 100);
        }
        printf("\n src: %lld, nodes_visited: %lld, edges_visited: %lld",
               (long long) src, (long long) nodes_visited, (long long) edges_visited);
        if (total_queued > 0)
        {
            printf(", total queued: %lld", total_queued);
        }
        if (redundant_work > 0)
        {
            printf(", redundant work: %.2f%%", redundant_work);
        }
        printf("\n");
    }
}

/******************************************************************************
 * BFS Testing Routines
 *****************************************************************************/

/**
 * @brief A simple CPU-based reference BFS ranking implementation.
 *
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 *
 * @param[in] graph Reference to the CSR graph we process on
 * @param[in] source_path Host-side vector to store CPU computed labels for each node
 * @param[in] predecessor Host-side vector to store CPU computed predecessor for each node
 * @param[in] src Source node where BFS starts
 */
template<
    typename VertexId,
    typename Value,
    typename SizeT,
    bool MARK_PREDECESSORS,
    bool ENABLE_IDEMPOTENCE>
void SimpleReferenceBfs(
    const Csr<VertexId, Value, SizeT>       *graph,
    VertexId                                *source_path,
    VertexId                                *predecessor,
    VertexId                                src)
{
    //initialize distances
    for (VertexId i = 0; i < graph->nodes; ++i) {
        source_path[i] = ENABLE_IDEMPOTENCE? -1: util::MaxValue<VertexId>()-1;
        if (MARK_PREDECESSORS)
            predecessor[i] = -1;
    }
    source_path[src] = 0;
    VertexId search_depth = 0;

    // Initialize queue for managing previously-discovered nodes
    std::deque<VertexId> frontier;
    frontier.push_back(src);

    //
    // Perform BFS
    //

    CpuTimer cpu_timer;
    cpu_timer.Start();
    while (!frontier.empty())
    {
        // Dequeue node from frontier
        VertexId dequeued_node = frontier.front();
        frontier.pop_front();
        VertexId neighbor_dist = source_path[dequeued_node] + 1;

        // Locate adjacency list
        SizeT edges_begin = graph->row_offsets[dequeued_node];
        SizeT edges_end = graph->row_offsets[dequeued_node + 1];

        for (SizeT edge = edges_begin; edge < edges_end; ++edge) {
            //Lookup neighbor and enqueue if undiscovered
            VertexId neighbor = graph->column_indices[edge];
            if (source_path[neighbor] > neighbor_dist || source_path[neighbor] == -1) {
                source_path[neighbor] = neighbor_dist;
                if (MARK_PREDECESSORS)
                    predecessor[neighbor] = dequeued_node;
                if (search_depth < neighbor_dist)
                {
                    search_depth = neighbor_dist;
                }
                frontier.push_back(neighbor);
            }
        }
    }

    if (MARK_PREDECESSORS)
        predecessor[src] = -1;

    cpu_timer.Stop();
    float elapsed = cpu_timer.ElapsedMillis();
    search_depth++;

    printf("CPU BFS finished in %lf msec. cpu_search_depth: %d\n",
           elapsed, search_depth);
}

/**
 * @brief Run BFS tests
 *
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 * @tparam INSTRUMENT
 * @tparam MARK_PREDECESSORS
 *
 * @param[in] graph Reference to the CSR graph we process on
 * @param[in] src Source node where BFS starts
 * @param[in] max_grid_size Maximum CTA occupancy
 * @param[in] num_gpus Number of GPUs
 * @param[in] max_queue_sizing Scaling factor used in edge mapping
 * @param[in] iterations Number of iterations for running the test
 * @param[in] traversal_mode Graph traversal mode: Load-balanced or Dynamic cooperative
 * @param[in] context CudaContext pointer for moderngpu APIs
 *
 */
template <
    typename    VertexId,
    typename    Value,
    typename    SizeT,
    bool        INSTRUMENT,
    bool        DEBUG,
    bool        SIZE_CHECK,
    bool        MARK_PREDECESSORS,
    bool        ENABLE_IDEMPOTENCE>
void RunTests(Test_Parameter *parameter)
{
    typedef BFSProblem<
        VertexId,
        SizeT,
        Value,
        MARK_PREDECESSORS,
        ENABLE_IDEMPOTENCE,
        (MARK_PREDECESSORS && ENABLE_IDEMPOTENCE)> 
    BfsProblem; // does not use double buffer

    typedef BFSEnactor<BfsProblem, 
        INSTRUMENT, 
        DEBUG, 
        SIZE_CHECK>
    BfsEnactor;

    Csr<VertexId, Value, SizeT>
                 *graph                 = (Csr<VertexId, Value, SizeT>*)parameter->graph;
    VertexId      src                   = (VertexId)parameter -> src;
    int           max_grid_size         = parameter -> max_grid_size;
    int           num_gpus              = parameter -> num_gpus;
    double        max_queue_sizing      = parameter -> max_queue_sizing;
    double        max_queue_sizing1     = parameter -> max_queue_sizing1;
    double        max_in_sizing         = parameter -> max_in_sizing;
    ContextPtr   *context               = (ContextPtr*)parameter -> context;
    std::string   partition_method      = parameter -> partition_method;
    int          *gpu_idx               = parameter -> gpu_idx;
    hipStream_t *streams               = parameter -> streams;
    float         partition_factor      = parameter -> partition_factor;
    int           partition_seed        = parameter -> partition_seed;
    bool          g_quick               = parameter -> g_quick;
    bool          g_stream_from_host    = parameter -> g_stream_from_host;
    int           traversal_mode        = parameter -> traversal_mode;
    SizeT         iterations            = parameter -> iterations;
    size_t       *org_size              = new size_t  [num_gpus];
    // Allocate host-side label array (for both reference and gpu-computed results)
    VertexId     *reference_labels      = new VertexId[graph->nodes];
    VertexId     *reference_preds       = new VertexId[graph->nodes];
    VertexId     *h_labels              = new VertexId[graph->nodes];
    VertexId     *reference_check_label = (g_quick) ? NULL : reference_labels;
    VertexId     *reference_check_preds = NULL;
    VertexId     *h_preds               = NULL;

    if (MARK_PREDECESSORS) {
        h_preds = new VertexId[graph->nodes];
        if (!g_quick) {
              reference_check_preds = reference_preds;
        }            
    }
 
    for (int gpu=0;gpu<num_gpus;gpu++)
    {
        size_t dummy;
        hipSetDevice(gpu_idx[gpu]);
        hipMemGetInfo(&(org_size[gpu]),&dummy);
    }
    // Allocate BFS enactor map
    BfsEnactor *enactor= new BfsEnactor(num_gpus, gpu_idx);
            
    // Allocate problem on GPU
    BfsProblem *problem = new BfsProblem;
    util::GRError(problem->Init(
        g_stream_from_host,
        graph,
        NULL,
        num_gpus,
        gpu_idx,
        partition_method,
        streams,
        max_queue_sizing,
        max_in_sizing,
        partition_factor,
        partition_seed), "Problem BFS Initialization Failed", __FILE__, __LINE__);
    util::GRError(enactor->Init (context, problem, max_grid_size, traversal_mode), "BFS Enactor init failed", __FILE__, __LINE__);

    //
    // Compute reference CPU BFS solution for source-distance
    //
    if (reference_check_label != NULL)
    {
        printf("Computing reference value ...\n");
        SimpleReferenceBfs<VertexId, Value, SizeT, MARK_PREDECESSORS, ENABLE_IDEMPOTENCE>(
            graph,
            reference_check_label,
            reference_check_preds,
            src);
        printf("\n");
    }

    Stats     *stats       = new Stats("GPU BFS");
    long long total_queued = 0;
    VertexId  search_depth = 0;
    double    avg_duty     = 0.0; 
    float     elapsed      = 0.0;

    // Perform BFS
    CpuTimer cpu_timer;

    for (int iter = 0; iter < iterations; ++iter)
    {
        util::GRError(problem->Reset(src, enactor->GetFrontierType(), max_queue_sizing, max_queue_sizing1), "BFS Problem Data Reset Failed", __FILE__, __LINE__);
        util::GRError(enactor->Reset(), "BFS Enactor Reset failed", __FILE__, __LINE__);

        util::GRError("Error before Enact", __FILE__, __LINE__);
        printf("__________________________\n");fflush(stdout);
        cpu_timer.Start();
        util::GRError(enactor->Enact(src, traversal_mode), "BFS Problem Enact Failed", __FILE__, __LINE__);
        cpu_timer.Stop();
        printf("--------------------------\n");fflush(stdout);
        elapsed += cpu_timer.ElapsedMillis();
    }

    elapsed /= iterations;

    enactor->GetStatistics(total_queued, search_depth, avg_duty);

    // Copy out results
    util::GRError(problem->Extract(h_labels, h_preds), "BFS Problem Data Extraction Failed", __FILE__, __LINE__);

    // Verify the result
    if (reference_check_label != NULL) {
        if (!ENABLE_IDEMPOTENCE) {
            printf("Label Validity: ");
            int error_num = CompareResults(h_labels, reference_check_label, graph->nodes, true);
            if (error_num > 0)
                printf("%d errors occurred.\n", error_num);
        } else {
            if (!MARK_PREDECESSORS) {
                printf("Label Validity: ");
                int error_num = CompareResults(h_labels, reference_check_label, graph->nodes, true);
                if (error_num > 0)
                    printf("%d errors occurred.\n", error_num);
            }
        }
    }

    printf("\nFirst 40 labels of the GPU result."); 
    // Display Solution
    DisplaySolution<VertexId, SizeT, MARK_PREDECESSORS, ENABLE_IDEMPOTENCE>
        (h_labels, h_preds, graph->nodes);

    DisplayStats<MARK_PREDECESSORS>(
        *stats,
        src,
        h_labels,
        graph,
        elapsed,
        search_depth,
        total_queued,
        avg_duty);

    printf("\n\tMemory Usage(B)\t");
    for (int gpu=0;gpu<num_gpus;gpu++)
    if (num_gpus>1) {if (gpu!=0) printf(" #keys%d,0\t #keys%d,1\t #ins%d,0\t #ins%d,1",gpu,gpu,gpu,gpu); else printf(" #keys%d,0\t #keys%d,1", gpu, gpu);}
    else printf(" #keys%d,0\t #keys%d,1", gpu, gpu);
    if (num_gpus>1) printf(" #keys%d",num_gpus);
    printf("\n");
    double max_queue_sizing_[2] = {0,0}, max_in_sizing_=0;
    for (int gpu=0;gpu<num_gpus;gpu++)
    {   
        size_t gpu_free,dummy;
        hipSetDevice(gpu_idx[gpu]);
        hipMemGetInfo(&gpu_free,&dummy);
        printf("GPU_%d\t %ld",gpu_idx[gpu],org_size[gpu]-gpu_free);
        for (int i=0;i<num_gpus;i++)
        {  
            for (int j=0; j<2; j++)
            { 
                SizeT x=problem->data_slices[gpu]->frontier_queues[i].keys[j].GetSize();
                printf("\t %lld", (long long) x); 
                double factor = 1.0*x/(num_gpus>1?problem->graph_slices[gpu]->in_counter[i]:problem->graph_slices[gpu]->nodes);
                if (factor > max_queue_sizing_[j]) max_queue_sizing_[j]=factor;
            }
            if (num_gpus>1 && i!=0 )
            for (int t=0;t<2;t++)
            {   
                SizeT x=problem->data_slices[gpu][0].keys_in[t][i].GetSize();
                printf("\t %lld", (long long) x); 
                double factor = 1.0*x/problem->graph_slices[gpu]->in_counter[i];
                if (factor > max_in_sizing_) max_in_sizing_=factor;
            }   
        }   
        if (num_gpus>1) printf("\t %lld", (long long)(problem->data_slices[gpu]->frontier_queues[num_gpus].keys[0].GetSize()));
        printf("\n");
    }   
    printf("\t queue_sizing =\t %lf \t %lf", max_queue_sizing_[0], max_queue_sizing_[1]);
    if (num_gpus>1) printf("\t in_sizing =\t %lf", max_in_sizing_);
    printf("\n");

    // Cleanup
    if (org_size        ) {delete[] org_size        ; org_size         = NULL;}
    if (stats           ) {delete   stats           ; stats            = NULL;}
    if (enactor         ) {delete   enactor         ; enactor          = NULL;}
    if (problem         ) {delete   problem         ; problem          = NULL;}
    if (reference_labels) {delete[] reference_labels; reference_labels = NULL;}
    if (reference_preds ) {delete[] reference_preds ; reference_preds  = NULL;}
    if (h_labels        ) {delete[] h_labels        ; h_labels         = NULL;}
    if (h_preds         ) {delete[] h_preds         ; h_preds          = NULL;}

    //hipDeviceSynchronize();
}

template <
    typename    VertexId,
    typename    Value,
    typename    SizeT,
    bool        INSTRUMENT,
    bool        DEBUG,
    bool        SIZE_CHECK,
    bool        MARK_PREDECESSORS>
void RunTests_enable_idempotence(Test_Parameter *parameter)
{
    if (parameter->enable_idempotence) RunTests
        <VertexId, Value, SizeT, INSTRUMENT, DEBUG, SIZE_CHECK, MARK_PREDECESSORS, 
        true > (parameter);
   else RunTests
        <VertexId, Value, SizeT, INSTRUMENT, DEBUG, SIZE_CHECK, MARK_PREDECESSORS,
        false> (parameter);
}

template <
    typename    VertexId,
    typename    Value,
    typename    SizeT,
    bool        INSTRUMENT,
    bool        DEBUG,
    bool        SIZE_CHECK>
void RunTests_mark_predecessors(Test_Parameter *parameter)
{
    if (parameter->mark_predecessors) RunTests_enable_idempotence
        <VertexId, Value, SizeT, INSTRUMENT, DEBUG, SIZE_CHECK,
        true > (parameter);
   else RunTests_enable_idempotence
        <VertexId, Value, SizeT, INSTRUMENT, DEBUG, SIZE_CHECK, 
        false> (parameter);
}

template <
    typename      VertexId,
    typename      Value,
    typename      SizeT,
    bool          INSTRUMENT,
    bool          DEBUG>
void RunTests_size_check(Test_Parameter *parameter)
{
    if (parameter->size_check) RunTests_mark_predecessors
        <VertexId, Value, SizeT, INSTRUMENT, DEBUG, 
        true > (parameter);
   else RunTests_mark_predecessors
        <VertexId, Value, SizeT, INSTRUMENT, DEBUG, 
        false> (parameter);
}

template <
    typename    VertexId,
    typename    Value,
    typename    SizeT,
    bool        INSTRUMENT>
void RunTests_debug(Test_Parameter *parameter)
{
    if (parameter->debug) RunTests_size_check
        <VertexId, Value, SizeT, INSTRUMENT, 
        true > (parameter);
    else RunTests_size_check
        <VertexId, Value, SizeT, INSTRUMENT, 
        false> (parameter);
}

template <
    typename      VertexId,
    typename      Value,
    typename      SizeT>
void RunTests_instrumented(Test_Parameter *parameter)
{
    if (parameter->instrumented) RunTests_debug
        <VertexId, Value, SizeT, 
        true > (parameter);
    else RunTests_debug
        <VertexId, Value, SizeT, 
        false> (parameter);
}

/**
 * @brief RunTests entry
 *
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 *
 * @param[in] graph Reference to the CSR graph we process on
 * @param[in] args Reference to the command line arguments
 * @param[in] context CudaContext pointer for moderngpu APIs
 */
template <
    typename VertexId,
    typename Value,
    typename SizeT>
void RunTests(
    Csr<VertexId, Value, SizeT> *graph,
    CommandLineArgs             &args,
    int                          num_gpus,
    ContextPtr                  *context,
    int                         *gpu_idx,
    hipStream_t                *streams)
{
    string src_str="";
    Test_Parameter *parameter = new Test_Parameter;   
 
    parameter -> Init(args);
    parameter -> graph              = graph;
    parameter -> num_gpus           = num_gpus;
    parameter -> context            = context;
    parameter -> gpu_idx            = gpu_idx;
    parameter -> streams            = streams;

    args.GetCmdLineArgument("src", src_str);
    if (src_str.empty()) {
        parameter->src = 0;
    } else if (src_str.compare("randomize") == 0) {
        parameter->src = graphio::RandomNode(graph->nodes);
    } else if (src_str.compare("largestdegree") == 0) {
        int temp;
        parameter->src = graph->GetNodeWithHighestDegree(temp);
    } else {
        args.GetCmdLineArgument("src", parameter->src);
    }
    printf("src = %lld\n", (long long) parameter->src);

    // traversal mode
    args.GetCmdLineArgument("traversal-mode", parameter->traversal_mode);
    if (parameter->traversal_mode == -1)
    {
        parameter->traversal_mode = graph->GetAverageDegree() > 8 ? 0 : 1;
    }

    RunTests_instrumented<VertexId, Value, SizeT>(parameter);
}

/******************************************************************************
* Main
******************************************************************************/

int main( int argc, char** argv)
{
    CommandLineArgs  args(argc, argv);
    int              num_gpus     = 0;
    int             *gpu_idx      = NULL;
    ContextPtr      *context      = NULL;
    hipStream_t    *streams      = NULL;
    bool             g_undirected = false;

    if ((argc < 2) || (args.CheckCmdLineFlag("help"))) {
        Usage();
        return 1;
    }

    if (args.CheckCmdLineFlag  ("device"))
    {   
        std::vector<int> gpus;
        args.GetCmdLineArguments<int>("device",gpus);
        num_gpus   = gpus.size();
        gpu_idx    = new int[num_gpus];
        for (int i=0;i<num_gpus;i++) 
            gpu_idx[i] = gpus[i];
    } else {
        num_gpus   = 1;
        gpu_idx    = new int[num_gpus];
        gpu_idx[0] = 0;
    }
    streams  = new hipStream_t[num_gpus * num_gpus *2];
    context  = new ContextPtr  [num_gpus * num_gpus];
    printf("Using %d gpus: ", num_gpus);
    for (int gpu=0;gpu<num_gpus;gpu++) 
    {
        printf(" %d ", gpu_idx[gpu]);
        util::SetDevice(gpu_idx[gpu]);
        for (int i=0;i<num_gpus*2;i++)
        {
            int _i=gpu*num_gpus*2+i;
            util::GRError(hipStreamCreate(&streams[_i]), "hipStreamCreate fialed.",__FILE__,__LINE__);
            if (i<num_gpus) context[gpu*num_gpus+i] = mgpu::CreateCudaDeviceAttachStream(gpu_idx[gpu],streams[_i]);
        }
    }
    printf("\n"); fflush(stdout);
    
    // Parse graph-contruction params
    g_undirected = args.CheckCmdLineFlag("undirected");

    std::string graph_type = argv[1];
    int flags = args.ParsedArgc();
    int graph_args = argc - flags - 1;

    if (graph_args < 1) {
        Usage();
        return 1;
    }

    //
    // Construct graph and perform search(es)
    //

#if USE64

    typedef long long int VertexId;
    typedef long long int Value;
    typedef long long int SizeT;

#else

    typedef int VertexId;                   // Use as the node identifier
    typedef int Value;                      // Use as the value type
    typedef int SizeT;                      // Use as the graph size type

#endif

    Csr<VertexId, Value, SizeT> csr(false); // default for stream_from_host
    if (graph_args < 1) { Usage(); return 1; }

    if (graph_type == "market")
    {
        // Matrix-market coordinate-formatted graph file
        if (graph_args < 1) { Usage(); return 1; }

        char *market_filename = (graph_args == 2) ? argv[2] : NULL;
        if (graphio::BuildMarketGraph<false>(
            market_filename, 
            csr, 
            g_undirected,
            false) != 0) // no inverse graph
        {
            return 1;
        }
    } else if (graph_type == "rmat")
    {
        // parse rmat parameters
        SizeT rmat_nodes = 1 << 10;
        SizeT rmat_edges = 1 << 10;
        SizeT rmat_scale = 10;
        SizeT rmat_edgefactor = 48;
        double rmat_a = 0.57;
        double rmat_b = 0.19;
        double rmat_c = 0.19;
        double rmat_d = 1-(rmat_a+rmat_b+rmat_c);
        int    rmat_seed = -1;

        args.GetCmdLineArgument("rmat_scale", rmat_scale);
        rmat_nodes = 1 << rmat_scale;
        args.GetCmdLineArgument("rmat_nodes", rmat_nodes);
        args.GetCmdLineArgument("rmat_edgefactor", rmat_edgefactor);
        rmat_edges = rmat_nodes * rmat_edgefactor;
        args.GetCmdLineArgument("rmat_edges", rmat_edges);
        args.GetCmdLineArgument("rmat_a", rmat_a);
        args.GetCmdLineArgument("rmat_b", rmat_b);
        args.GetCmdLineArgument("rmat_c", rmat_c);
        rmat_d = 1-(rmat_a+rmat_b+rmat_c);
        args.GetCmdLineArgument("rmat_d", rmat_d);
        args.GetCmdLineArgument("rmat_seed", rmat_seed);

        CpuTimer cpu_timer;
        cpu_timer.Start();
        if (graphio::BuildRmatGraph<false>(
                rmat_nodes,
                rmat_edges,
                csr,
                g_undirected,
                rmat_a,
                rmat_b,
                rmat_c,
                rmat_d,
                1,
                1,
                rmat_seed) != 0)
        {
            return 1;
        }
        cpu_timer.Stop();
        float elapsed = cpu_timer.ElapsedMillis();
        printf("graph generated: %.3f ms, a = %.3f, b = %.3f, c = %.3f, d = %.3f\n", elapsed, rmat_a, rmat_b, rmat_c, rmat_d);
    } else if (graph_type == "rgg") {
        
        SizeT rgg_nodes = 1 << 10;
        SizeT rgg_scale = 10;
        double rgg_thfactor  = 0.55;
        double rgg_threshold = rgg_thfactor * sqrt(log(rgg_nodes) / rgg_nodes);
        double rgg_vmultipiler = 1;
        int    rgg_seed        = -1;
        
        args.GetCmdLineArgument("rgg_scale", rgg_scale);
        rgg_nodes = 1 << rgg_scale;
        args.GetCmdLineArgument("rgg_nodes", rgg_nodes);
        args.GetCmdLineArgument("rgg_thfactor", rgg_thfactor);
        rgg_threshold = rgg_thfactor * sqrt(log(rgg_nodes) / rgg_nodes);
        args.GetCmdLineArgument("rgg_threshold", rgg_threshold);
        args.GetCmdLineArgument("rgg_vmultipiler", rgg_vmultipiler);
        args.GetCmdLineArgument("rgg_seed", rgg_seed);

        CpuTimer cpu_timer;
        cpu_timer.Start();
        if (graphio::BuildRggGraph<false>(
            rgg_nodes,
            csr,
            rgg_threshold,
            g_undirected,
            rgg_vmultipiler,
            1,
            rgg_seed) !=0)
        {
            return 1;
        }
        cpu_timer.Stop();
        float elapsed = cpu_timer.ElapsedMillis();
        printf("graph generated: %.3f ms, threshold = %.3lf, vmultipiler = %.3lf\n", elapsed, rgg_threshold, rgg_vmultipiler);
    }else
    {
        fprintf(stderr, "Unspecified graph type\n");
        return 1;
    }

    csr.PrintHistogram();
    RunTests(&csr, args, num_gpus, context, gpu_idx, streams);

    return 0;
}

