#include "hip/hip_runtime.h"
// ----------------------------------------------------------------
// Gunrock -- Fast and Efficient GPU Graph Library
// ----------------------------------------------------------------
// This source code is distributed under the terms of LICENSE.TXT
// in the root directory of this source distribution.
// ----------------------------------------------------------------

/**
 * @file
 * test_sssp.cu
 *
 * @brief Simple test driver program for single source shorest path.
 */

#include <stdio.h>
#include <string>
#include <deque>
#include <vector>
#include <iostream>

// Utilities and correctness-checking
#include <gunrock/util/test_utils.cuh>

// Graph construction utils
#include <gunrock/graphio/market.cuh>
#include <gunrock/graphio/rmat.cuh>
#include <gunrock/graphio/rgg.cuh>

// SSSP includes
#include <gunrock/app/sssp/sssp_enactor.cuh>
#include <gunrock/app/sssp/sssp_problem.cuh>
#include <gunrock/app/sssp/sssp_functor.cuh>

// Operator includes
#include <gunrock/oprtr/advance/kernel.cuh>
#include <gunrock/oprtr/filter/kernel.cuh>
#include <gunrock/priority_queue/kernel.cuh>

#include <moderngpu.cuh>

// Boost includes for CPU dijkstra SSSP reference algorithms
#include <boost/config.hpp>
#include <boost/graph/graph_traits.hpp>
#include <boost/graph/adjacency_list.hpp>
#include <boost/graph/dijkstra_shortest_paths.hpp>
#include <boost/property_map/property_map.hpp>

using namespace gunrock;
using namespace gunrock::util;
using namespace gunrock::oprtr;
using namespace gunrock::app::sssp;


/******************************************************************************
 * Defines, constants, globals
 ******************************************************************************/

//bool g_verbose;
//bool g_undirected;
//bool g_quick;
//bool g_stream_from_host;

/******************************************************************************
 * Housekeeping Routines
 ******************************************************************************/
void Usage()
{
    printf(
        " test_sssp <graph type> <graph type args> [--device=<device_index>]\n"
        " [--undirected] [--instrumented] [--src=<source index>] [--quick=<0|1>]\n"
        " [--mark-pred] [--queue-sizing=<scale factor>] [--traversal-mode=<0|1>]\n"
        " [--in-sizing=<in/out queue scale factor>] [--disable-size-check]\n"
        " [--grid-size=<grid size>] [partition_method=<random|biasrandom|clustered|metis>]\n"
        " [--v] [--iteration-num=<num>]\n"
        "\n"
        "Graph types and args:\n"
        "  market [<file>]\n"
        "    Reads a Matrix-Market coordinate-formatted graph of directed / undirected\n"
        "    edges from stdin (or from the optionally-specified file).\n"
        "  --device=<device_index>   Set GPU device for running the test. [Default: 0].\n"
        "  --undirected              Treat the graph as undirected (symmetric).\n"
        "  --instrumented            Keep kernels statics [Default: Disable].\n"
        "                            total_queued, search_depth and barrier duty\n"
        "                            (a relative indicator of load imbalance.)\n"
        "  --src=<source vertex id>  Begins SSSP from the source [Default: 0].\n"
        "                            If randomize: from a random source vertex.\n"
        "                            If largestdegree: from largest degree vertex.\n"
        "  --quick=<0 or 1>          Skip the CPU validation: 1, or not: 0 [Default: 1].\n"
        "  --mark-pred               Keep both label info and predecessor info.\n"
        "  --queue-sizing=<factor>   Allocates a frontier queue sized at:\n"
        "                            (graph-edges * <scale factor>) [Default: 1.0].\n"
        "  --v                       Print verbose per iteration debug info.\n"
        "  --iteration-num=<number>  Number of runs to perform the test [Default: 1].\n"
        "  --traversal-mode=<0 or 1> Set traversal strategy, 0 for Load-Balanced,\n"
        "                            1 for Dynamic-Cooperative [Default: dynamic\n"
        "                            determine based on average degree].\n"
        );
}

/**
 * @brief Displays the SSSP result (i.e., distance from source)
 *
 * @param[in] source_path Search depth from the source for each node.
 * @param[in] nodes Number of nodes in the graph.
 */
template<typename VertexId, typename SizeT>
void DisplaySolution (VertexId *source_path, SizeT num_nodes)
{
    if (num_nodes > 40) num_nodes = 40;

    printf("[");
    for (VertexId i = 0; i < num_nodes; ++i)
    {
        PrintValue(i);
        printf(":");
        PrintValue(source_path[i]);
        printf(" ");
    }
    printf("]\n");
}

/**
 * Performance/Evaluation statistics
 */

struct Stats {
    const char *name;
    Statistic rate;
    Statistic search_depth;
    Statistic redundant_work;
    Statistic duty;

    Stats() : name(NULL), rate(), search_depth(), redundant_work(), duty() {}
    Stats(const char *name) : name(name), rate(), search_depth(), redundant_work(), duty() {}
};

struct Test_Parameter : gunrock::app::TestParameter_Base {
public:
    //bool          mark_predecessors ;// Whether or not to mark src-distance vs. parent vertices
    int delta_factor;
    double max_queue_sizing1;

    Test_Parameter()
    { 
        delta_factor = 16;
        mark_predecessors = false;
        max_queue_sizing1 = -1.0;
    }   

    ~Test_Parameter()
    {   
    }   

    void Init(CommandLineArgs &args)
    {   
        TestParameter_Base::Init(args);
        mark_predecessors = args.CheckCmdLineFlag("mark-pred");
        args.GetCmdLineArgument("delta-factor"    , delta_factor    );
        args.GetCmdLineArgument("queue-sizing1", max_queue_sizing1);
    }   
};

/**
 * @brief Displays timing and correctness statistics
 *
 * @tparam MARK_PREDECESSORS
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 *
 * @param[in] stats Reference to the Stats object defined in RunTests
 * @param[in] src Source node where SSSP starts
 * @param[in] h_labels Host-side vector stores computed labels for validation
 * @param[in] graph Reference to the CSR graph we process on
 * @param[in] elapsed Total elapsed kernel running time
 * @param[in] search_depth Maximum search depth of the SSSP algorithm
 * @param[in] total_queued Total element queued in SSSP kernel running process
 * @param[in] avg_duty Average duty of the SSSP kernels
 */
template<
    typename VertexId,
    typename Value,
    typename SizeT>
void DisplayStats(
    Stats               &stats,
    VertexId            src,
    Value               *h_labels,
    const Csr<VertexId, Value, SizeT> &graph,
    double              elapsed,
    VertexId            search_depth,
    long long           total_queued,
    double              avg_duty)
{
    // Compute nodes and edges visited
    SizeT edges_visited = 0;
    SizeT nodes_visited = 0;
    for (VertexId i = 0; i < graph.nodes; ++i) {
        if (h_labels[i] < util::MaxValue<VertexId>()) {
            ++nodes_visited;
            edges_visited += graph.row_offsets[i+1] - graph.row_offsets[i];
        }
    }

    double redundant_work = 0.0;
    if (total_queued > 0)
    {
        redundant_work =
            ((double) total_queued - edges_visited) / edges_visited;
    }
    redundant_work *= 100;

    // Display test name
    printf("[%s] finished.", stats.name);

    // Display statistics
    if (nodes_visited < 5)
    {
        printf("Fewer than 5 vertices visited.\n");
    }
    else
    {
        // Display the specific sample statistics
        double m_teps = (double) edges_visited / (elapsed * 1000.0);
        printf("\n elapsed: %.4f ms, rate: %.4f MiEdges/s", elapsed, m_teps);
        if (search_depth != 0)
            printf(", search_depth: %lld", (long long) search_depth);
        printf("\n src: %lld, nodes_visited: %lld, edges_visited: %lld",
               (long long) src, (long long) nodes_visited, (long long) edges_visited);
        if (avg_duty != 0)
        {
            printf("\n avg CTA duty: %.2f%%", avg_duty * 100);
        }
        if (total_queued > 0)
        {
            printf(", total queued: %lld", total_queued);
        }
        if (redundant_work > 0)
        {
            printf(", redundant work: %.2f%%", redundant_work);
        }
        printf("\n");
    }
}

/******************************************************************************
 * SSSP Testing Routines
 *****************************************************************************/

/**
 * @brief A simple CPU-based reference SSSP ranking implementation.
 *
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 * @tparam MARK_PREDECESSORS
 *
 * @param[in] graph Reference to the CSR graph we process on
 * @param[in] node_values Host-side vector to store CPU computed labels for each node
 * @param[in] node_preds Host-side vector to store CPU computed predecessors for each node
 * @param[in] src Source node where SSSP starts
 */
template<
    typename VertexId,
    typename Value,
    typename SizeT,
    bool     MARK_PREDECESSORS>
void SimpleReferenceSssp(
    const Csr<VertexId, Value, SizeT>       &graph,
    Value                                   *node_values,
    VertexId                                *node_preds,
    VertexId                                src)
{
    using namespace boost;

    // Prepare Boost Datatype and Data structure
    typedef adjacency_list<vecS, vecS, directedS, no_property,
                           property <edge_weight_t, unsigned int> > Graph;

    typedef graph_traits<Graph>::vertex_descriptor vertex_descriptor;
    typedef graph_traits<Graph>::edge_descriptor edge_descriptor;

    typedef std::pair<unsigned int, unsigned int> Edge;

    Edge* edges = (Edge*)malloc(sizeof(Edge)*graph.edges);
    unsigned int *weight =
        (unsigned int*)malloc(sizeof(unsigned int)*graph.edges);

    for (int i = 0; i < graph.nodes; ++i)
    {
        for (int j = graph.row_offsets[i]; j < graph.row_offsets[i+1]; ++j)
        {
            edges[j] = Edge(i, graph.column_indices[j]);
            weight[j] = graph.edge_values[j];
        }
    }

    Graph g(edges, edges + graph.edges, weight, graph.nodes);

    std::vector<unsigned int> d(graph.nodes);
    std::vector<vertex_descriptor> p(graph.nodes);
    vertex_descriptor s = vertex(src, g);

    property_map<Graph, vertex_index_t>::type indexmap = get(vertex_index, g);

    //
    // Perform SSSP
    //

    CpuTimer cpu_timer;
    cpu_timer.Start();

    if (MARK_PREDECESSORS)
        dijkstra_shortest_paths(
            g, s,
            predecessor_map(boost::make_iterator_property_map(p.begin(), get(boost::vertex_index, g))).
            distance_map(boost::make_iterator_property_map(d.begin(), get(boost::vertex_index, g))));
    else
        dijkstra_shortest_paths(
            g, s,
            distance_map(boost::make_iterator_property_map(d.begin(), get(boost::vertex_index, g))));
    cpu_timer.Stop();
    float elapsed = cpu_timer.ElapsedMillis();

    printf("CPU SSSP finished in %lf msec.\n", elapsed);

    Coo<unsigned int, unsigned int>* sort_dist = NULL;
    Coo<unsigned int, unsigned int>* sort_pred = NULL;
    sort_dist = (Coo<unsigned int, unsigned int>*)malloc(
        sizeof(Coo<unsigned int, unsigned int>) * graph.nodes);
    if (MARK_PREDECESSORS)
        sort_pred = (Coo<unsigned int, unsigned int>*)malloc(
            sizeof(Coo<unsigned int, unsigned int>) * graph.nodes);

    graph_traits < Graph >::vertex_iterator vi, vend;
    for (tie(vi, vend) = vertices(g); vi != vend; ++vi)
    {
        sort_dist[(*vi)].row = (*vi);
        sort_dist[(*vi)].col = d[(*vi)];
    }
    std::stable_sort(
        sort_dist, sort_dist + graph.nodes,
        RowFirstTupleCompare<Coo<unsigned int, unsigned int> >);

    if (MARK_PREDECESSORS)
    {
        for (tie(vi, vend) = vertices(g); vi != vend; ++vi)
        {
            sort_pred[(*vi)].row = (*vi);
            sort_pred[(*vi)].col = p[(*vi)];
        }
        std::stable_sort(
            sort_pred, sort_pred + graph.nodes,
            RowFirstTupleCompare<Coo<unsigned int, unsigned int> >);
    }

    for (int i = 0; i < graph.nodes; ++i)
    {
        node_values[i] = sort_dist[i].col;
    }
    if (MARK_PREDECESSORS)
        for (int i = 0; i < graph.nodes; ++i)
        {
            node_preds[i] = sort_pred[i].col;
        }

    free(sort_dist);
    if (MARK_PREDECESSORS) free(sort_pred);
}


/**
 * @brief Run SSSP tests
 *
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 * @tparam INSTRUMENT
 * @tparam MARK_PREDECESSORS
 *
 * @param[in] graph Reference to the CSR graph we process on
 * @param[in] src Source node where SSSP starts
 * @param[in] max_grid_size Maximum CTA occupancy
 * @param[in] queue_sizing Scaling factor used in edge mapping
 * @param[in] num_gpus Number of GPUs
 * @param[in] delta_factor Parameter to specify delta in delta-stepping SSSP
 * @param[in] iterations Number of iteration for running the test
 & @param[in] traversal_mode Load-balanced or Dynamic cooperative
 * @param[in] context CudaContext pointer for moderngpu APIs
 */
template <
    typename VertexId,
    typename Value,
    typename SizeT,
    bool INSTRUMENT,
    bool DEBUG,
    bool SIZE_CHECK,
    bool MARK_PREDECESSORS>
void RunTests(Test_Parameter *parameter)
{
    typedef SSSPProblem<
        VertexId,
        SizeT,
        Value,
        MARK_PREDECESSORS> Problem;

    typedef SSSPEnactor<
        Problem,
        INSTRUMENT,
        DEBUG,
        SIZE_CHECK> Enactor;

    Csr<VertexId, Value, SizeT>
                 *graph                 = (Csr<VertexId, Value, SizeT>*)parameter->graph;
    VertexId      src                   = (VertexId)parameter -> src;
    int           max_grid_size         = parameter -> max_grid_size;
    int           num_gpus              = parameter -> num_gpus;
    double        max_queue_sizing      = parameter -> max_queue_sizing;
    double        max_in_sizing         = parameter -> max_in_sizing;
    ContextPtr   *context               = (ContextPtr*)parameter -> context;
    std::string   partition_method      = parameter -> partition_method;
    int          *gpu_idx               = parameter -> gpu_idx;
    hipStream_t *streams               = parameter -> streams;
    float         partition_factor      = parameter -> partition_factor;
    int           partition_seed        = parameter -> partition_seed;
    bool          g_quick               = parameter -> g_quick;
    bool          g_stream_from_host    = parameter -> g_stream_from_host;
    int           delta_factor          = parameter -> delta_factor;
    int           iterations            = parameter -> iterations;
    int           traversal_mode        = parameter -> traversal_mode;
    size_t       *org_size              = new size_t[num_gpus];
    // Allocate host-side label array (for both reference and gpu-computed results)
    Value        *reference_labels      = new Value[graph->nodes];
    Value        *h_labels              = new Value[graph->nodes];
    Value        *reference_check_label = (g_quick) ? NULL : reference_labels;
    VertexId     *reference_preds       = MARK_PREDECESSORS ? new VertexId[graph->nodes] : NULL;
    VertexId     *h_preds               = MARK_PREDECESSORS ? new VertexId[graph->nodes] : NULL;
    VertexId     *reference_check_pred  = (g_quick || !MARK_PREDECESSORS) ? NULL : reference_preds;

    for (int gpu=0;gpu<num_gpus;gpu++)
    {
        size_t dummy;
        hipSetDevice(gpu_idx[gpu]);
        hipMemGetInfo(&(org_size[gpu]),&dummy);
    }
        
    // Allocate SSSP enactor map
    Enactor* enactor = new Enactor(num_gpus, gpu_idx);

    // Allocate problem on GPU
    Problem *problem = new Problem;
    util::GRError(problem->Init(
        g_stream_from_host,
        graph,
        NULL,
        num_gpus,
        gpu_idx,
        partition_method,
        streams,
        delta_factor,
        max_queue_sizing,
        max_in_sizing,
        partition_factor,
        partition_seed), "Problem SSSP Initialization Failed", __FILE__, __LINE__);
    util::GRError(enactor->Init (context, problem, max_grid_size, traversal_mode), "SSSP Enactor init failed", __FILE__, __LINE__);
    //
    // Compute reference CPU SSSP solution for source-distance
    //
    if (reference_check_label != NULL)
    {
        printf("Computing reference value ...\n");
        SimpleReferenceSssp<VertexId, Value, SizeT, MARK_PREDECESSORS>(
                *graph,
                reference_check_label,
                reference_check_pred,
                src);
        printf("\n");
    }

    Stats      *stats       = new Stats("GPU SSSP");
    long long  total_queued = 0;
    VertexId   search_depth = 0;
    double     avg_duty     = 0.0;
    float      elapsed      = 0.0f;

    // Perform SSSP
    CpuTimer cpu_timer;

    for (int iter = 0; iter < iterations; ++iter)
    {
        util::GRError(problem->Reset(src, enactor->GetFrontierType(), max_queue_sizing), "SSSP Problem Data Reset Failed", __FILE__, __LINE__); 
        util::GRError(enactor->Reset(), "SSSP Enactor Reset failed", __FILE__, __LINE__);

        printf("__________________________\n");fflush(stdout);
        cpu_timer.Start();
        util::GRError(enactor->Enact(src, traversal_mode), "SSSP Problem Enact Failed", __FILE__, __LINE__);
        cpu_timer.Stop();
        printf("--------------------------\n");fflush(stdout);
        elapsed += cpu_timer.ElapsedMillis();
    }
    elapsed /= iterations;

    enactor->GetStatistics(total_queued, search_depth, avg_duty);

    // Copy out results
    util::GRError(problem->Extract(h_labels, h_preds), "SSSP Problem Data Extraction Failed", __FILE__, __LINE__);

    for (SizeT i=0; i<graph->nodes;i++)
    if (reference_check_label[i]==-1) reference_check_label[i]=util::MaxValue<Value>();

    // Display Solution
    printf("\nFirst 40 labels of the GPU result.\n"); 
    DisplaySolution(h_labels, graph->nodes);
 
    // Verify the result
    if (reference_check_label != NULL) {
        printf("Label Validity: ");
        int error_num = CompareResults(h_labels, reference_check_label, graph->nodes, true);
        if (error_num > 0)
            printf("%d errors occurred.\n", error_num);
        printf("\nFirst 40 labels of the reference CPU result.\n"); 
        DisplaySolution(reference_check_label, graph->nodes);
    }
    
    if (MARK_PREDECESSORS) {
        printf("\nFirst 40 preds of the GPU result.\n"); 
        DisplaySolution(h_preds, graph->nodes);
        if (reference_check_label != NULL) 
        {
            printf("\nFirst 40 preds of the reference CPU result (could be different because the paths are not unique).\n"); 
            DisplaySolution(reference_check_pred, graph->nodes);
        }
    }

    DisplayStats(
        *stats,
        src,
        h_labels,
        *graph,
        elapsed,
        search_depth,
        total_queued,
        avg_duty);

    printf("\n\tMemory Usage(B)\t");
    for (int gpu=0;gpu<num_gpus;gpu++)
    if (num_gpus>1) {if (gpu!=0) printf(" #keys%d,0\t #keys%d,1\t #ins%d,0\t #ins%d,1",gpu,gpu,gpu,gpu); else printf(" #keys%d,0\t #keys%d,1", gpu, gpu);}
    else printf(" #keys%d,0\t #keys%d,1", gpu, gpu);
    if (num_gpus>1) printf(" #keys%d",num_gpus);
    printf("\n");
    double max_queue_sizing_[2] = {0,0}, max_in_sizing_=0;
    for (int gpu=0;gpu<num_gpus;gpu++)
    {   
        size_t gpu_free,dummy;
        hipSetDevice(gpu_idx[gpu]);
        hipMemGetInfo(&gpu_free,&dummy);
        printf("GPU_%d\t %ld",gpu_idx[gpu],org_size[gpu]-gpu_free);
        for (int i=0;i<num_gpus;i++)
        {   
            for (int j=0; j<2; j++)
            {   
                SizeT x=problem->data_slices[gpu]->frontier_queues[i].keys[j].GetSize();
                printf("\t %lld", (long long) x); 
                double factor = 1.0*x/(num_gpus>1?problem->graph_slices[gpu]->in_counter[i]:problem->graph_slices[gpu]->nodes);
                if (factor > max_queue_sizing_[j]) max_queue_sizing_[j]=factor;
            }   
            if (num_gpus>1 && i!=0 )
            for (int t=0;t<2;t++)
            {   
                SizeT x=problem->data_slices[gpu][0].keys_in[t][i].GetSize();
                printf("\t %lld", (long long) x); 
                double factor = 1.0*x/problem->graph_slices[gpu]->in_counter[i];
                if (factor > max_in_sizing_) max_in_sizing_=factor;
            }   
        }   
        if (num_gpus>1) printf("\t %lld", (long long)(problem->data_slices[gpu]->frontier_queues[num_gpus].keys[0].GetSize()));
        printf("\n");
    }   
    printf("\t queue_sizing =\t %lf \t %lf", max_queue_sizing_[0], max_queue_sizing_[1]);
    if (num_gpus>1) printf("\t in_sizing =\t %lf", max_in_sizing_);
    printf("\n");

    // Cleanup
    if (org_size        ) {delete[] org_size        ; org_size         = NULL;}
    if (stats           ) {delete   stats           ; stats            = NULL;}
    if (enactor         ) {delete   enactor         ; enactor          = NULL;}
    if (problem         ) {delete   problem         ; problem          = NULL;}
    if (reference_labels) {delete[] reference_labels; reference_labels = NULL;}
    if (h_labels        ) {delete[] h_labels        ; h_labels         = NULL;}
    if (reference_preds ) {delete[] reference_preds ; reference_preds  = NULL;}
    if (h_preds         ) {delete[] h_preds         ; h_preds          = NULL;}

    //hipDeviceSynchronize();
}

template <
    typename    VertexId,
    typename    Value,
    typename    SizeT,
    bool        INSTRUMENT,
    bool        DEBUG,
    bool        SIZE_CHECK>
void RunTests_mark_predecessors(Test_Parameter *parameter)
{
    if (parameter->mark_predecessors) RunTests
        <VertexId, Value, SizeT, INSTRUMENT, DEBUG, SIZE_CHECK,
        true > (parameter);
   else RunTests
        <VertexId, Value, SizeT, INSTRUMENT, DEBUG, SIZE_CHECK,
        false> (parameter);
}

template <
    typename      VertexId,
    typename      Value,
    typename      SizeT,
    bool          INSTRUMENT,
    bool          DEBUG>
void RunTests_size_check(Test_Parameter *parameter)
{
    if (parameter->size_check) RunTests_mark_predecessors
        <VertexId, Value, SizeT, INSTRUMENT, DEBUG,
        true > (parameter);
   else RunTests_mark_predecessors
        <VertexId, Value, SizeT, INSTRUMENT, DEBUG,
        false> (parameter);
}

template <
    typename    VertexId,
    typename    Value,
    typename    SizeT,
    bool        INSTRUMENT>
void RunTests_debug(Test_Parameter *parameter)
{
    if (parameter->debug) RunTests_size_check
        <VertexId, Value, SizeT, INSTRUMENT,
        true > (parameter);
    else RunTests_size_check
        <VertexId, Value, SizeT, INSTRUMENT,
        false> (parameter);
}

template <
    typename      VertexId,
    typename      Value,
    typename      SizeT>
void RunTests_instrumented(Test_Parameter *parameter)
{
    if (parameter->instrumented) RunTests_debug
        <VertexId, Value, SizeT,
        true > (parameter);
    else RunTests_debug
        <VertexId, Value, SizeT,
        false> (parameter);
}

/**
 * @brief RunTests entry
 *
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 *
 * @param[in] graph Reference to the CSR graph we process on
 * @param[in] args Reference to the command line arguments
 * @param[in] context CudaContext pointer for moderngpu APIs
 */
template <
    typename VertexId,
    typename Value,
    typename SizeT>
void RunTests(
    Csr<VertexId, Value, SizeT> *graph,
    CommandLineArgs             &args,
    int                         num_gpus,
    ContextPtr                  *context,
    int                         *gpu_idx,
    hipStream_t                *streams)
{
    string src_str = "";
    Test_Parameter *parameter = new Test_Parameter;
    
    parameter -> Init(args);
    parameter -> graph              = graph;
    parameter -> num_gpus           = num_gpus;
    parameter -> context            = context;
    parameter -> gpu_idx            = gpu_idx;
    parameter -> streams            = streams;

    args.GetCmdLineArgument("src", src_str);
    if (src_str.empty()) {
        parameter->src = 0;
    } else if (src_str.compare("randomize") == 0) {
        parameter->src = graphio::RandomNode(graph->nodes);
    } else if (src_str.compare("largestdegree") == 0) {
        int max_degree;
        parameter->src = graph->GetNodeWithHighestDegree(max_degree);
        printf("Using highest degree (%d) vertex: %d\n", max_degree, parameter->src);
    } else {
        args.GetCmdLineArgument("src", parameter->src);
    }

    // traversal mode
    args.GetCmdLineArgument("traversal-mode", parameter->traversal_mode);
    if (parameter->traversal_mode == -1)
    {
        parameter->traversal_mode = graph->GetAverageDegree() > 8 ? 0 : 1;
    }

    printf("src = %lld\n", parameter->src);

    RunTests_instrumented<VertexId, Value, SizeT>(parameter);
}

/******************************************************************************
* Main
******************************************************************************/

int main( int argc, char** argv)
{
    CommandLineArgs args(argc, argv);
    int          num_gpus = 0;
    int          *gpu_idx = NULL;
    ContextPtr   *context = NULL;
    hipStream_t *streams = NULL;
    bool          g_undirected = false;

    if ((argc < 2) || (args.CheckCmdLineFlag("help"))) {
        Usage();
        return 1;
    }

    if (args.CheckCmdLineFlag  ("device"))
    {
        std::vector<int> gpus;
        args.GetCmdLineArguments<int>("device",gpus);
        num_gpus   = gpus.size();
        gpu_idx    = new int[num_gpus];
        for (int i=0;i<num_gpus;i++)
            gpu_idx[i] = gpus[i];
    } else {
        num_gpus   = 1;
        gpu_idx    = new int[num_gpus];
        gpu_idx[0] = 0;
    }
    streams  = new hipStream_t[num_gpus * num_gpus *2];
    context  = new ContextPtr  [num_gpus * num_gpus];
    printf("Using %d gpus: ", num_gpus);
    for (int gpu=0;gpu<num_gpus;gpu++)
    {
        printf(" %d ", gpu_idx[gpu]);
        util::SetDevice(gpu_idx[gpu]);
        for (int i=0;i<num_gpus*2;i++)
        {
            int _i=gpu*num_gpus*2+i;
            util::GRError(hipStreamCreate(&streams[_i]), "hipStreamCreate fialed.",__FILE__,__LINE__);
            if (i<num_gpus) context[gpu*num_gpus+i] = mgpu::CreateCudaDeviceAttachStream(gpu_idx[gpu],streams[_i]);
        }
    }
    printf("\n"); fflush(stdout);
    
    // Parse graph-contruction params
    g_undirected = args.CheckCmdLineFlag("undirected");
    std::string graph_type = argv[1];
    int flags = args.ParsedArgc();
    int graph_args = argc - flags - 1;

    if (graph_args < 1) {
        Usage();
        return 1;
    }
	
    //
    // Construct graph and perform search(es)
    //
    typedef int VertexId;                   // Use as the node identifier type
    typedef int Value;                      // Use as the value type
    typedef long long  int SizeT;                      // Use as the graph size type
    Csr<VertexId, Value, SizeT> csr(false); // default value for stream_from_host is false
    if (graph_args < 1) { Usage(); return 1; }

    if (graph_type == "market") {
    // Matrix-market coordinate-formatted graph file

        char *market_filename = (graph_args == 2) ? argv[2] : NULL;
        if (graphio::BuildMarketGraph<true>(
            market_filename, 
            csr, 
            g_undirected,
            false) != 0) // no inverse graph
        {
            return 1;
        }

    } else if (graph_type == "rmat")
    {   
        // parse rmat parameters
        SizeT rmat_nodes = 1 << 10; 
        SizeT rmat_edges = 1 << 10; 
        SizeT rmat_scale = 10; 
        SizeT rmat_edgefactor = 48; 
        double rmat_a = 0.57;
        double rmat_b = 0.19;
        double rmat_c = 0.19;
        double rmat_d = 1-(rmat_a+rmat_b+rmat_c);
        double rmat_vmultipiler = 20;
        double rmat_vmin        = 1;
        int    rmat_seed        = -1;

        args.GetCmdLineArgument("rmat_scale", rmat_scale);
        rmat_nodes = 1 << rmat_scale;
        args.GetCmdLineArgument("rmat_nodes", rmat_nodes);
        args.GetCmdLineArgument("rmat_edgefactor", rmat_edgefactor);
        rmat_edges = rmat_nodes * rmat_edgefactor;
        rmat_vmultipiler = rmat_edgefactor * 2;
        args.GetCmdLineArgument("rmat_edges", rmat_edges);
        args.GetCmdLineArgument("rmat_a", rmat_a);
        args.GetCmdLineArgument("rmat_b", rmat_b);
        args.GetCmdLineArgument("rmat_c", rmat_c);
        rmat_d = 1-(rmat_a+rmat_b+rmat_c);
        args.GetCmdLineArgument("rmat_d", rmat_d);
        args.GetCmdLineArgument("rmat_vmultipiler", rmat_vmultipiler);
        args.GetCmdLineArgument("rmat_vmin", rmat_vmin);
        args.GetCmdLineArgument("rmat_seed", rmat_seed);

        CpuTimer cpu_timer;
        cpu_timer.Start();
        if (graphio::BuildRmatGraph<true>(
                rmat_nodes,
                rmat_edges,
                csr,
                g_undirected,
                rmat_a,
                rmat_b,
                rmat_c,
                rmat_d,
                rmat_vmultipiler,
                rmat_vmin,
                rmat_seed) != 0)
        {   
            return 1;
        }   
        cpu_timer.Stop();
        float elapsed = cpu_timer.ElapsedMillis();
        printf("graph generated: %.3f ms, a = %.3f, b = %.3f, c = %.3f, d = %.3f\n", elapsed, rmat_a, rmat_b, rmat_c, rmat_d);
    } else if (graph_type == "rgg") {
    
        SizeT rgg_nodes = 1 << 10; 
        SizeT rgg_scale = 10; 
        double rgg_thfactor  = 0.55;
        double rgg_threshold = rgg_thfactor * sqrt(log(rgg_nodes) / rgg_nodes);
        double rgg_vmultipiler = 20;
        double rgg_vmin = 1;
        int    rgg_seed = -1;
    
        args.GetCmdLineArgument("rgg_scale", rgg_scale);
        rgg_nodes = 1 << rgg_scale;
        args.GetCmdLineArgument("rgg_nodes", rgg_nodes);
        args.GetCmdLineArgument("rgg_thfactor", rgg_thfactor);
        rgg_threshold = rgg_thfactor * sqrt(log(rgg_nodes) / rgg_nodes);
        args.GetCmdLineArgument("rgg_threshold", rgg_threshold);
        args.GetCmdLineArgument("rgg_vmultipiler", rgg_vmultipiler);
        args.GetCmdLineArgument("rgg_vmin", rgg_vmin);
        args.GetCmdLineArgument("rgg_seed", rgg_seed);

        CpuTimer cpu_timer;
        cpu_timer.Start();
        if (graphio::BuildRggGraph<true>(
            rgg_nodes,
            csr,
            rgg_threshold,
            g_undirected,
            rgg_vmultipiler,
            rgg_vmin,
            rgg_seed) !=0)
        {
            return 1;
        }
        cpu_timer.Stop();
        float elapsed = cpu_timer.ElapsedMillis();
        printf("graph generated: %.3f ms, threshold = %.3lf, vmultipiler = %.3lf\n", elapsed, rgg_threshold, rgg_vmultipiler);
    } else {
        // Unknown graph type
        fprintf(stderr, "Unspecified graph type\n");
        return 1;
    }

    csr.PrintHistogram();
    csr.DisplayGraph(true); //print graph with edge_value
    //util::cpu_mt::PrintCPUArray("row_offsets", csr.row_offsets,csr.nodes+1);
    //util::cpu_mt::PrintCPUArray("colum_indiece", csr.column_indices, csr.edges);
    
    csr.GetAverageEdgeValue();
    csr.GetAverageDegree();
    int max_degree;
    csr.GetNodeWithHighestDegree(max_degree);
    printf("max degree:%d\n", max_degree);

    // Run tests
    RunTests(&csr, args, num_gpus, context, gpu_idx, streams);

    return 0;
}
